/**
 * Copyright 2013 Axel Huebl, Felix Schmitt, Heiko Burau, Rene Widera
 *
 * This file is part of PIConGPU.
 *
 * PIConGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * PIConGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with PIConGPU.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @mainpage PIConGPU-Frame
 *
 * Project with HZDR for porting their PiC-code to a GPU cluster.
 *
 * \image html picongpu.jpg
 *
 * @author Heiko Burau, Rene Widera, Wolfgang Hoenig, Felix Schmitt, Axel Huebl, Michael Bussmann, Guido Juckeland
 */

//#include <thread>
#include <pthread.h>
#include "nn.h"
#include "pipeline.h"

pthread_mutex_t mutex1, mutex2;
pthread_cond_t condition1, condition2;
bool P_to_T = false, T_to_P = false;

extern pthread_mutex_t mutex1, mutex2;
extern pthread_cond_t condition1, condition2;
extern bool P_to_T, T_to_P;

extern int sockpull_t, sockpush_t, sockpull_p, sockpush_p;

//extern hipIpcMemHandle_t *g_mem_handle;

// include the heap with the arguments given in the config
#include "mallocMC/mallocMC_utils.hpp"

// basic files for mallocMC
#include "mallocMC/mallocMC_overwrites.hpp"
#include "mallocMC/mallocMC_hostclass.hpp"

// Load all available policies for mallocMC
#include "mallocMC/CreationPolicies.hpp"
#include "mallocMC/DistributionPolicies.hpp"
#include "mallocMC/OOMPolicies.hpp"
#include "mallocMC/ReservePoolPolicies.hpp"
#include "mallocMC/AlignmentPolicies.hpp"

// configurate the CreationPolicy "Scatter"
struct ScatterConfig
{
    /* 2MiB page can hold around 256 particle frames */
    typedef boost::mpl::int_<2*1024*1024> pagesize;
    /* accessblocks, regionsize and wastefactor are not finale selected
       and might be performance sensitive*/
    typedef boost::mpl::int_<4> accessblocks;
    typedef boost::mpl::int_<8> regionsize;
    typedef boost::mpl::int_<2> wastefactor;
    /* resetfreedpages is used to minimize memory fracmentation while different
       frame sizes were used*/
    typedef boost::mpl::bool_<true> resetfreedpages;
};

// Define a new allocator and call it ScatterAllocator
// which resembles the behaviour of ScatterAlloc
typedef mallocMC::Allocator<
mallocMC::CreationPolicies::Scatter<ScatterConfig>,
mallocMC::DistributionPolicies::Noop,
mallocMC::OOMPolicies::ReturnNull,
mallocMC::ReservePoolPolicies::SimpleCudaMalloc,
mallocMC::AlignmentPolicies::Shrink<>
> ScatterAllocator;

//use ScatterAllocator to replace malloc/free
MALLOCMC_SET_ALLOCATOR_TYPE( ScatterAllocator );

#include <simulation_defines.hpp>
#include <mpi.h>
#include "ArgsParser.hpp"
#include "communication/manager_common.h"

#include <cupti.h>
#include <string>
#include <sstream>
#include <fstream>
#include <sys/mman.h>

using namespace PMacc;
using namespace picongpu;

/*! start of PIConGPU
 *
 * @param argc count of arguments in argv
 * @param argv arguments of program start
 */

#define CUPTI_CALL(call)                                                \                             
  do {                                                                  \                             
    CUptiResult _status = call;                                         \                             
    if (_status != CUPTI_SUCCESS) {                                     \
      const char *errstr;                                               \                             
      cuptiGetResultString(_status, &errstr);                           \                             
      std::cerr << __FILE__ << ":" << __LINE__ << ":" << "error: function " << #call  << "failed with error " << errstr << std::endl; \
                       \
      std::exit(-1);                                                         \                        
    }                                                                   \                             
  } while (0)

#define BUF_SIZE (32 * 1024 * 1024)
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \                      
  (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

void CUPTIAPI take_buffer(uint8_t **buffer, size_t *size, size_t *max_num_records)
{
  uint8_t *bfr = (uint8_t *) malloc(BUF_SIZE + ALIGN_SIZE);
  if (bfr == NULL) {
    std::cerr << "Error: out of memory" << std::endl;
    std::exit(-1);
  }

  *size = BUF_SIZE;
  *buffer = ALIGN_BUFFER(bfr, ALIGN_SIZE);
  *max_num_records = 0;
}

long g_last_end = 0, g_last_end_api = 0; 
int g_rank = -1;
std::ofstream g_ofs, g_ofs_api;

void CUPTIAPI return_buffer(hipCtx_t ctx, uint32_t stream_id, uint8_t *buffer, size_t size, size_t valid_size)
{
  CUptiResult status;
  CUpti_Activity *record = NULL;
  int num_kernel_records=0, num_api_records=0;
  if (valid_size > 0) {
    do {
      status = cuptiActivityGetNextRecord(buffer, valid_size, &record);
      if (status == CUPTI_SUCCESS) {
         //num_records++;
         if((record->kind == CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL)||(record->kind == CUPTI_ACTIVITY_KIND_KERNEL))
         { 
            num_kernel_records++;
            CUpti_ActivityKernel2 *kernel = (CUpti_ActivityKernel2 *) record;
            //if(g_last_end==0)
            //{
               //g_last_end = kernel->start; //kernel->start;
            //}
	    g_ofs << kernel->start << " " << kernel->end << " " << kernel->correlationId << /*" " << kernel->end - kernel->start << " " << kernel->start - g_last_end <<*/ std::endl;
            /*
            if((long)(kernel->start) >= g_last_end){
               g_ofs << kernel->start << " " << kernel->end << " " << kernel->correlationId << " " << kernel->end - kernel->start << " " << kernel->start - g_last_end << std::endl;
               g_last_end = kernel->end;
            }
            else if((long)(kernel->end) >= g_last_end)
            {
               g_last_end = kernel->end;
            }*/
         }
         if((record->kind == CUPTI_ACTIVITY_KIND_RUNTIME)||(record->kind == CUPTI_ACTIVITY_KIND_DRIVER))
         {
             num_api_records++;
             CUpti_ActivityAPI *api = (CUpti_ActivityAPI *) record;
             /*if(g_last_end_api==0)
             {
                g_last_end_api = api->start;
             }*/
             //g_ofs_api << api->start << " " << api->correlationId << std::endl;
         }
      }
      else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED)
        break;
      else {
        CUPTI_CALL(status);
      }
    } while (1);
    //g_ofs << "dump " << num_kernel_records << std::endl;
    //g_ofs_api << "dump " << num_api_records << std::endl;
    // report any records dropped from the queue
    size_t dropped;
    CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, stream_id, &dropped));
    if (dropped != 0) {
      std::cerr << "Dropped " << (unsigned int) dropped << "activity records" << std::endl;
    }

  }

  free(buffer);
}

namespace patch
{
    template < typename T > std::string to_string( const T& n )
    {
        std::ostringstream stm ;
        stm << n ;
        return stm.str() ;
    }
}

int sockpull_t, sockpush_t, sockpull_p, sockpush_p;

void* some_thrust_method(void *data)
{
    /*pthread_mutex_lock(&mutex1);
    while(!P_to_T)
    {
      pthread_cond_wait(&condition1, &mutex1);
    }
    std::cout << "Thrust received signal\n";
    P_to_T = false;
    pthread_mutex_unlock(&mutex1);

    pthread_mutex_lock(&mutex2);
    std::cout << "Thrust sending signal\n";
    T_to_P = true;
    pthread_mutex_unlock(&mutex2);
    pthread_cond_signal(&condition2);*/
    sockpull_t = nn_socket (AF_SP, NN_PULL);
    assert (sockpull_t >= 0);
    std::cerr << "[t]t_to_p socket ok" << std::endl;
    assert (nn_bind (sockpull_t, "ipc:///tmp/t_to_p.ipc") >= 0);
    std::cerr << "t_to_p bind ok" << std::endl;
    sockpush_t = nn_socket (AF_SP, NN_PUSH);
    assert (sockpush_t >= 0);
    std::cerr << "[t]p_to_t socket ok" << std::endl;
    assert (nn_connect (sockpush_t, "ipc:///tmp/p_to_t.ipc") >= 0);
    std::cerr << "p_to_t connect ok" << std::endl;

    while(1)
    {
    while (1)
    {
      char *buf = NULL;
      int bytes = nn_recv (sockpull_t, &buf, NN_MSG, 0);
      assert (bytes >= 0);
      std::cerr << "[T] : " << buf << std::endl;
      nn_freemsg (buf);
      break;
    } 
    
    const char * msg = "T to P";
    int bytes = nn_send (sockpush_t, msg, strlen (msg) + 1, 0);
    assert (bytes == strlen (msg) + 1);
    std::cerr << "[T] T to P" << std::endl;
    }
    return NULL;
}

MPI_Comm MPI_COMM_WORLD_INSITU;
hipIpcMemHandle_t *g_mem_handle=NULL;

int main(int argc, char **argv)
{
    MPI_CHECK(MPI_Init(&argc, &argv));

    MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &g_rank));
    //g_mem_handle = (hipIpcMemHandle_t *)mmap(NULL, sizeof(hipIpcMemHandle_t), PROT_READ | PROT_WRITE, MAP_SHARED | MAP_ANONYMOUS, 0, 0);

    //MPI_CHECK(MPI_Comm_split(MPI_COMM_WORLD, g_rank%2, g_rank, &MPI_COMM_WORLD_INSITU));
    pthread_t some_thrust_method_handle;
    if(pthread_create(&some_thrust_method_handle, NULL, some_thrust_method, NULL))
    {
       std::cout << "Error calling pthread_create" << std::endl;
    }
    sockpush_p = nn_socket (AF_SP, NN_PUSH);
    assert (sockpush_p >= 0);
    std::cerr << "[p]t_to_p socket ok" << std::endl;
    assert (nn_connect (sockpush_p, "ipc:///tmp/t_to_p.ipc") >= 0);
    std::cerr << "t_to_p connect ok" << std::endl;
    sockpull_p = nn_socket (AF_SP, NN_PULL);
    assert (sockpull_p >= 0);
    std::cerr << "[p]p_to_t socket ok" << std::endl;
    assert (nn_bind (sockpull_p, "ipc:///tmp/p_to_t.ipc") >= 0);
    std::cerr << "p_to_t bind ok" << std::endl;

    int insitu_rank=-1;
    //MPI_CHECK(MPI_Comm_rank(MPI_COMM_WORLD, &insitu_rank));

    //if(g_rank % 2 == 0)
    //{
    std::string filename_k(patch::to_string(g_rank)+"_kernel_picongpu.out");
    std::string filename_rd(patch::to_string(g_rank)+"_api_picongpu.out");
    g_ofs.open(filename_k.c_str());
    g_ofs_api.open(filename_rd.c_str());
    //}
    //else
    //{
    /*
    std::string filename_k(patch::to_string(g_rank)+"_kernel_thrust.out");
    std::string filename_rd(patch::to_string(g_rank)+"_api_thrust.out");
    g_ofs.open(filename_k.c_str());
    g_ofs_api.open(filename_rd.c_str());
    */
    //}
    cuptiActivityRegisterCallbacks(take_buffer, return_buffer);
    //cuptiActivityEnable(CUPTI_ACTIVITY_KIND_RUNTIME);
    //cuptiActivityEnable(CUPTI_ACTIVITY_KIND_DRIVER);
    cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL);

    int errorCode = 1;

    //if(g_rank % 2 == 0)
    //{
        //std::cout << "picongpu " << insitu_rank << " " << g_rank << std::endl;
    picongpu::simulation_starter::SimStarter sim;
    ArgsParser::ArgsErrorCode parserCode = sim.parseConfigs(argc, argv);

    /*if (!sim.parseConfigs(argc, argv))
    {
        MPI_CHECK(MPI_Finalize());
        return 1;
    }

    sim.load();
    sim.start();
    sim.unload();*/

    switch(parserCode)
    {
        case ArgsParser::ERROR:
            errorCode = 1;
            break;
        case ArgsParser::SUCCESS:
            sim.load();
            sim.start();
            sim.unload();
            /*set error code to valid (1) after the simulation terminates*/
        case ArgsParser::SUCCESS_EXIT:
            errorCode = 0;
            break;
    };
    //}
    //else
    //{
        //thrust
        /*
        std::cout << "thrust " << insitu_rank << " " << g_rank << std::endl;
        while(g_mem_handle == NULL){}
        std::cerr << "main::hipIpcOpenMemHandle mem_handle " << g_mem_handle << " " << (*g_mem_handle) << std::endl;
        void *d_ptr=NULL;
        hipError_t err = hipIpcOpenMemHandle((void **) &d_ptr, *g_mem_handle, hipIpcMemLazyEnablePeerAccess);
        if(err != hipSuccess)
        {
            std::cerr << err << " hipErrorMapFailed-" << hipErrorMapFailed << " hipErrorInvalidHandle-" << hipErrorInvalidHandle << " cudaErrorTooManyPeers-" << cudaErrorTooManyPeers << std::endl;
        }
        else
        {
            std::cerr << "main::hipIpcOpenMemHandle d_ptr " << d_ptr << std::endl;
        }
        */
    //}

    //cuptiActivityDisable(CUPTI_ACTIVITY_KIND_RUNTIME);
    //cuptiActivityDisable(CUPTI_ACTIVITY_KIND_DRIVER);
    //cuptiActivityDisable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL);

    /*if(pthread_join(some_thrust_method_handle, NULL))
    {
        std::cout << "Error in pthread_join" << std::endl;
    }*/

    MPI_CHECK(MPI_Finalize());

    return 0;
}
